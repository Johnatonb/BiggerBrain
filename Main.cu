#include "hip/hip_runtime.h"
﻿#include <fstream>
#include "hip/hip_runtime.h"
#include ""
#include "BigBrain.hpp"
#include <stdio.h>

matrix<double> readImagesColumn(std::ifstream& file, int imageNum) {
    matrix<double> image(784, 1);
    file.seekg(16 + (784 * imageNum));
    for (size_t j = 0; j < 784; j++) {
        unsigned char temp;
        file.read((char*)&temp, sizeof(temp));
        image[j][0] = (double)temp / 255;
    }
    return image;
}


matrix<double> readImagesSquare(std::ifstream& file, int imageNum) {
    matrix<double> image(28, 28);
    file.seekg(16 + (784 * imageNum));
    for (size_t i = 0; i < 28; i++) {
        for (size_t j = 0; j < 28; j++) {
            unsigned char temp;
            file.read((char*)&temp, sizeof(temp));
            image[i][j] = (double)temp / 255;
        }
    }
    return image;
}

matrix<double> readLabel(std::ifstream& file, int imageNum) {
    matrix<double> label(10, 1);
    file.seekg(8 + imageNum);
    unsigned char temp;
    file.read((char*)&temp, sizeof(temp));
    //std::cout << (double)temp << std::endl;
    label[temp][0] = double(1);
    return label;
}


int main(){
	std::string trainingImages = R"(C:\Users\johnk\Downloads\train-images-idx3-ubyte\train-images.idx3-ubyte)";
	std::ifstream imageFile(trainingImages.c_str(), std::ios::binary);
	std::string trainingLabels = R"(C:\Users\johnk\Downloads\train-labels-idx1-ubyte\train-labels.idx1-ubyte)";
	std::ifstream labelFile(trainingLabels.c_str(), std::ios::binary);

	matrix<double> image = readImagesSquare(imageFile, 0);

	std::vector<matrix<double>> images;
	std::vector<matrix<double>> labels;

	for (size_t i = 0; i < 1000; i++) {
		images.push_back(readImagesColumn(imageFile, i));
		labels.push_back(readLabel(labelFile, i));
	}

	std::cout << "Finished Reading" << std::endl;

	bigBrain network(std::vector<size_t>{ 784, 16, 16, 10 });
	network.initialize();
	/*for (int i = 0; i < 100; i++) {
		network.learn(images, labels);
		std::cout << i << "%" << std::endl;
	}

	matrix<double> output = network.think(images[0]);
	for (int i = 0; i < 10; i++) {
		std::cout << output[i][0] << std::endl;
	}

	cv::imshow("my image", image.visualize());
	cv::waitKey(0);

	/*for (size_t i = 1; i <= 28; i++) {
		for (size_t j = 1; j <= 28; j++) {
			if (ceil(image[(i * j) - size_t(1)][0]) == 1) { std::cout << "."; }
			else { std::cout << " "; }
		}
		std::cout<<std::endl;
	}*/
}